
#include <hip/hip_runtime.h>
__global__ void kernel2(int m, int n, int k, double *d_A, double *d_B, double *d_C){
    int j = blockIdx.x*blockDim.x+threadIdx.x; 
    int i = blockIdx.y*blockDim.y+threadIdx.y; 

    if (i < m && j < n){
        double d_c1 = 0.0;
        for(int s = 0; s < k; s++){
            d_c1 += d_A[i*k + s] * d_B[s*n + j];
        }
        d_C[i*n + j] = d_c1;
    }
}

extern "C" { 
	void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C) { 

    
    double *d_A, *d_B, *d_C; //variable on device

    int size_matrix_A = m * k * sizeof(double);
    hipMalloc((void**)&d_A, size_matrix_A); // allocate memory on GPU
    int size_matrix_B = k * n * sizeof(double);
    hipMalloc((void**)&d_B, size_matrix_B);
    int size_matrix_C = m * n * sizeof(double);
    hipMalloc((void**)&d_C, size_matrix_C);

    //copy A and B to GPU
    hipMemcpy(d_A, A, size_matrix_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_matrix_B, hipMemcpyHostToDevice); 

    dim3 dimBlock(16,16,1);
    dim3 dimGrid((m + dimBlock.x-1)/dimBlock.x,(n + dimBlock.y-1)/dimBlock.y);

    kernel2<<<dimGrid,dimBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize(); 
    
    //transfer C back to CPU
    hipMemcpy(C, d_C, size_matrix_C, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C); }
    }