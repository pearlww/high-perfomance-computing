
#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" {
    void matmult_gpulib(int m, int n, int k, double *A, double *B, double *C) {
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0, beta = 0.0;
    int lda=k, ldb=n, ldc=n;
    
    double *d_A, *d_B, *d_C; //variable on device

    int size_matrix_A = m * k * sizeof(double);
    hipMalloc((void**)&d_A, size_matrix_A); // allocate memory on GPU
    int size_matrix_B = k * n * sizeof(double);
    hipMalloc((void**)&d_B, size_matrix_B);
    int size_matrix_C = m * n * sizeof(double);
    hipMalloc((void**)&d_C, size_matrix_C);

    //copy A and B to GPU
    hipMemcpy(d_A, A, size_matrix_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_matrix_B, hipMemcpyHostToDevice); 

    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, ldb, d_A, lda, &beta, d_C, ldc);
    hipDeviceSynchronize(); 
    
    //transfer C back to CPU
    hipMemcpy(C, d_C, size_matrix_C, hipMemcpyDeviceToHost);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C); 

}
}


    
