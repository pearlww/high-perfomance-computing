
#include <hip/hip_runtime.h>
#define STRIDE 2

__global__ void kernel3(int m, int n, int k, double *d_A, double *d_B, double *d_C){
    int i = (blockIdx.y * blockDim.y + threadIdx.y) * STRIDE; 
    int j = blockIdx.x * blockDim.x + threadIdx.x; 
    
    int sum1=0;
    for(int s1 = 0; s1 < STRIDE; s1++){
        sum1 = i + s1; 
        if (sum1 < m && j < n){
            d_C[sum1*n + j] = 0.0;
            for(int s = 0; s < k; s++){
                d_C[sum1*n + j] += d_A[sum1*k + s] * d_B[s*n + j];  }
        }   
    } 
}


extern "C" { 
	void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C) { 

    
    double *d_A, *d_B, *d_C; //variable on device

    int size_matrix_A = m * k * sizeof(double);
    hipMalloc((void**)&d_A, size_matrix_A); // allocate memory on GPU
    int size_matrix_B = k * n * sizeof(double);
    hipMalloc((void**)&d_B, size_matrix_B);
    int size_matrix_C = m * n * sizeof(double);
    hipMalloc((void**)&d_C, size_matrix_C);

    //copy A and B to GPU
    hipMemcpy(d_A, A, size_matrix_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_matrix_B, hipMemcpyHostToDevice); 

    dim3 dimBlock(16,16,1);
    dim3 dimGrid((m -1)/dimBlock.x+1,(n/STRIDE-1)/dimBlock.y+1) ; 

    kernel3<<<dimGrid,dimBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize(); 
    
    //transfer C back to CPU
    hipMemcpy(C, d_C, size_matrix_C, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C); }
    }